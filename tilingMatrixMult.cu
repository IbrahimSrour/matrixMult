#include "hip/hip_runtime.h"
%%writefile matrixMult.cu
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

const int MATRIX_SIZE = 2048;
const int SHARED_MEM_SIZE = 1024;

__global__ void matMul(const int *mat_a, const int *mat_b, int *mat_c) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  __shared__ int shared_a[SHARED_MEM_SIZE];
  __shared__ int shared_b[SHARED_MEM_SIZE];

  int temp = 0;

  for (int i = 0; i < MATRIX_SIZE; i += blockDim.x) {
    shared_a[threadIdx.y * blockDim.x + threadIdx.x] = mat_a[row * MATRIX_SIZE + i + threadIdx.x];
    shared_b[threadIdx.y * blockDim.x + threadIdx.x] = mat_b[i * MATRIX_SIZE + threadIdx.y * MATRIX_SIZE + col];

    __syncthreads();

    for (int j = 0; j < blockDim.x; j++) {
      temp += shared_a[threadIdx.y * blockDim.x + j] * shared_b[j * blockDim.x + threadIdx.x];
    }

    __syncthreads();
  }

  mat_c[row * MATRIX_SIZE + col] = temp;
}


int main() {
  
  size_t bytes = MATRIX_SIZE * MATRIX_SIZE * sizeof(int);

  vector<int> h_mat_a(MATRIX_SIZE * MATRIX_SIZE);
  vector<int> h_mat_b(MATRIX_SIZE * MATRIX_SIZE);
  vector<int> h_mat_c(MATRIX_SIZE * MATRIX_SIZE);

  generate(h_mat_a.begin(), h_mat_a.end(), []() { return rand() % 100; });
  generate(h_mat_b.begin(), h_mat_b.end(), []() { return rand() % 100; });

  int *d_mat_a, *d_mat_b, *d_mat_c;
  hipMalloc(&d_mat_a, bytes);
  hipMalloc(&d_mat_b, bytes);
  hipMalloc(&d_mat_c, bytes);
 hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start, 0);

hipMemcpy(d_mat_a, h_mat_a.data(), bytes, hipMemcpyHostToDevice);
hipMemcpy(d_mat_b, h_mat_b.data(), bytes, hipMemcpyHostToDevice);

int THREADS = 32;
int BLOCKS = MATRIX_SIZE / THREADS;

dim3 threads(THREADS, THREADS);
dim3 blocks(BLOCKS, BLOCKS);

matMul<<<blocks, threads>>>(d_mat_a, d_mat_b, d_mat_c);

hipMemcpy(h_mat_c.data(), d_mat_c, bytes, hipMemcpyDeviceToHost);

hipEventRecord(stop, 0);  // Move this line up here.
hipEventSynchronize(stop);

float elapsedTime;
hipEventElapsedTime(&elapsedTime, start, stop);

std::cout << "Elapsed time: " << elapsedTime << " ms\n";

hipEventDestroy(start);
hipEventDestroy(stop);

hipFree(d_mat_a);
hipFree(d_mat_b);
hipFree(d_mat_c);

return 0;
}
