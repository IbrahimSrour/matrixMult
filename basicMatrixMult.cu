#include "hip/hip_runtime.h"
%%writefile matrixMult.cu


#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

__global__ void matMulKernel(const int *matA, const int *matB, int *matC, int matSize) {
  
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  matC[row * matSize + col] = 0;
  
  for (int k = 0; k < matSize; k++) {
    
    matC[row * matSize + col] += matA[row * matSize + k] * matB[k * matSize + col];
  }
}

int main() {
  int matSize = 4096;

  size_t byteSize = matSize * matSize * sizeof(int);

  vector<int> hostA(matSize * matSize);
  vector<int> hostB(matSize * matSize);
  vector<int> hostC(matSize * matSize);

  generate(hostA.begin(), hostA.end(), []() { return rand() % 100; });
  generate(hostB.begin(), hostB.end(), []() { return rand() % 100; });

  int *devA, *devB, *devC;
  hipMalloc(&devA, byteSize);
  hipMalloc(&devB, byteSize);
  hipMalloc(&devC, byteSize);
hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  hipMemcpy(devA, hostA.data(), byteSize, hipMemcpyHostToDevice);
  hipMemcpy(devB, hostB.data(), byteSize, hipMemcpyHostToDevice);

  int THREADS = 32;
  int BLOCKS = matSize / THREADS;

  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  matMulKernel<<<blocks, threads>>>(devA, devB, devC, matSize);

  hipMemcpy(hostC.data(), devC, byteSize, hipMemcpyDeviceToHost);
hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);

  std::cout << "Elapsed time: " << elapsedTime << " ms\n";

  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(devA);
  hipFree(devB);
  hipFree(devC);

  return 0;
}
